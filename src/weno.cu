#include "Block3d_cuda.h"
#include "weno.h"


namespace block3d_cuda {

  // ---------------------------------------------------------------------------

  extern __constant__ Block3dInfo blk_info;
  
  // ---------------------------------------------------------------------------
  
  __device__ value_type weno5(const value_type Um2,
			      const value_type Um1,
			      const value_type U0,
			      const value_type Up1,
			      const value_type Up2) {

    static const value_type eps = 1.0e-6;
  
    value_type diff1 = Um2 - 2.0 * Um1 + U0;
    value_type diff2 = Um2 - 4.0 * Um1 + 3.0 * U0;
    value_type diff3 = Um1 - 2.0 * U0 + Up1;
    value_type diff4 = Um1 - Up1;
    value_type diff5 = U0 - 2.0 * Up1 + Up2;
    value_type diff6 = 3.0 * U0 - 4.0 * Up1 + Up2;

    value_type IS1 = (13.0/12.0) * diff1 * diff1 + 0.25 * diff2 * diff2 + eps;
    value_type IS2 = (13.0/12.0) * diff3 * diff3 + 0.25 * diff4 * diff4 + eps;
    value_type IS3 = (13.0/12.0) * diff5 * diff5 + 0.25 * diff6 * diff6 + eps;

    value_type w1 = 0.1 / (IS1 * IS1);   
    value_type w2 = 0.6 / (IS2 * IS2);   
    value_type w3 = 0.3 / (IS3 * IS3);   

    value_type sum = w1 + w2 + w3;
	  
    value_type f1 = (2.0 * Um2 - 7.0 * Um1 + 11.0 *U0) / 6.0;
    value_type f2 = (-Um1 + 5.0 * U0 + 2.0 *Up1) / 6.0;
    value_type f3 = (2.0 * U0 + 5.0 * Up1 - Up2) / 6.0;

    return (w1 * f1 + w2 * f2 + w3 * f3) / sum;
  
  }

  __device__ void rec_weno5(const value_type* f1, const value_type* f2,
			    const value_type* f3, const value_type* f4,
			    const value_type* f5,
			    const value_type R_l[constant::NEQ][constant::NEQ],
			    const value_type L_l[constant::NEQ][constant::NEQ],
			    value_type& rho_L,
			    value_type& u_L, value_type& v_L, value_type& w_L,
			    value_type& p_L) {
  
    value_type f_sten[constant::NEQ][5];
    value_type fc_sten[constant::NEQ][5];

    f_sten[0][0] = f1[0];
    f_sten[1][0] = f1[1];
    f_sten[2][0] = f1[2];
    f_sten[3][0] = f1[3];
    f_sten[4][0] = f1[4];

    f_sten[0][1] = f2[0];
    f_sten[1][1] = f2[1];
    f_sten[2][1] = f2[2];
    f_sten[3][1] = f2[3];
    f_sten[4][1] = f2[4];

    f_sten[0][2] = f3[0];
    f_sten[1][2] = f3[1];
    f_sten[2][2] = f3[2];
    f_sten[3][2] = f3[3];
    f_sten[4][2] = f3[4];

    f_sten[0][3] = f4[0];
    f_sten[1][3] = f4[1];
    f_sten[2][3] = f4[2];
    f_sten[3][3] = f4[3];
    f_sten[4][3] = f4[4];

    f_sten[0][4] = f5[0];
    f_sten[1][4] = f5[1];
    f_sten[2][4] = f5[2];
    f_sten[3][4] = f5[3];
    f_sten[4][4] = f5[4];

    for (size_type li = 0; li < constant::NEQ; li++) {
      for (size_type lj = 0; lj < 5; lj++) {

	value_type ss = 0.0;
	for (size_type lk = 0; lk < constant::NEQ; lk++) {
	  ss += L_l[li][lk] * f_sten[lk][lj];
	}
	fc_sten[li][lj] = ss;

      }
    }
 
    f_sten[0][0] = weno5(fc_sten[0][0], fc_sten[0][1], fc_sten[0][2], fc_sten[0][3], fc_sten[0][4]);
    f_sten[1][0] = weno5(fc_sten[1][0], fc_sten[1][1], fc_sten[1][2], fc_sten[1][3], fc_sten[1][4]);
    f_sten[2][0] = weno5(fc_sten[2][0], fc_sten[2][1], fc_sten[2][2], fc_sten[2][3], fc_sten[2][4]);
    f_sten[3][0] = weno5(fc_sten[3][0], fc_sten[3][1], fc_sten[3][2], fc_sten[3][3], fc_sten[3][4]);
    f_sten[4][0] = weno5(fc_sten[4][0], fc_sten[4][1], fc_sten[4][2], fc_sten[4][3], fc_sten[4][4]);

    for (size_type li = 0; li < constant::NEQ; li++) {
      value_type ss = 0.0;
      for (size_type lj = 0; lj < constant::NEQ; lj++) {
	ss += R_l[li][lj] * f_sten[lj][0];
      }
      f_sten[li][1] = ss;
    }

    rho_L = f_sten[0][1];
    u_L = f_sten[1][1] / rho_L;
    v_L = f_sten[2][1] / rho_L;
    w_L = f_sten[3][1] / rho_L;
    p_L = blk_info.gam1 * (f_sten[4][1] - 0.5 * rho_L * (u_L * u_L + v_L * v_L + w_L * w_L));
  
  }
  
}
