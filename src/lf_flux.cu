#include "hip/hip_runtime.h"
#include "Block3d_cuda.h"
#include "lf_flux.h"


namespace block3d_cuda {

  // ---------------------------------------------------------------------------

  extern __constant__ Block3dInfo blk_info;
  
  // ---------------------------------------------------------------------------
  
  __device__ void lf_flux(value_type rho_R,
			  value_type u_R, value_type v_R, value_type w_R,
			  value_type p_R,
			  value_type rho_L,
			  value_type u_L, value_type v_L, value_type w_L,
			  value_type p_L,
			  value_type nx, value_type ny, value_type nz,
			  value_type *flux) {

    const value_type gamma = blk_info.gamma;
    const value_type gam1 = blk_info.gam1;

    value_type c_R = std::sqrt(gamma * p_R / rho_R);
    value_type H_R = c_R * c_R / gam1 + 0.5 * (u_R * u_R + v_R * v_R + w_R * w_R);
    value_type rE_R = p_R / gam1 + 0.5 * rho_R * (u_R * u_R + v_R * v_R + w_R * w_R);

    value_type c_L = std::sqrt(gamma * p_L / rho_L);
    value_type H_L = c_L * c_L / gam1 + 0.5 * (u_L * u_L + v_L * v_L + w_L * w_L);
    value_type rE_L = p_L / gam1 + 0.5 * rho_L * (u_L * u_L + v_L * v_L + w_L * w_L);

    value_type q_R = u_R * nx + v_R * ny + w_R * nz;
    value_type q_L = u_L * nx + v_L * ny + w_L * nz;

    // ---------------------------------------------------------------------------
    value_type ws[3];
    value_type ws_max;
  
    ws[0] = std::abs(u_R) + c_R;
    ws_max = std::abs(u_L) + c_L;
    if (ws_max > ws[0]) ws[0] = ws_max;

    ws[1] = std::abs(v_R) + c_R;
    ws_max = std::abs(v_L) + c_L;
    if (ws_max > ws[1]) ws[1] = ws_max;

    ws[2] = std::abs(w_R) + c_R;
    ws_max = std::abs(w_L) + c_L;
    if (ws_max > ws[2]) ws[2] = ws_max;

    ws_max = std::abs(ws[0] * nx + ws[1] * ny + ws[2] * nz);
    // ---------------------------------------------------------------------------

    value_type diss[constant::NEQ];
    diss[0] = ws_max * (rho_R - rho_L);
    diss[1] = ws_max * (rho_R * u_R - rho_L * u_L);
    diss[2] = ws_max * (rho_R * v_R - rho_L * v_L);
    diss[3] = ws_max * (rho_R * w_R - rho_L * w_L);
    diss[4] = ws_max * (rE_R - rE_L);

    value_type f_L[constant::NEQ];
    f_L[0] = rho_L * q_L;
    f_L[1] = rho_L * q_L * u_L + p_L * nx;
    f_L[2] = rho_L * q_L * v_L + p_L * ny;
    f_L[3] = rho_L * q_L * w_L + p_L * nz;
    f_L[4] = rho_L * q_L * H_L;

    value_type f_R[constant::NEQ];
    f_R[0] = rho_R * q_R;
    f_R[1] = rho_R * q_R * u_R + p_R * nx;
    f_R[2] = rho_R * q_R * v_R + p_R * ny;
    f_R[3] = rho_R * q_R * w_R + p_R * nz;
    f_R[4] = rho_R * q_R * H_R;

    for (size_type li = 0; li < constant::NEQ; li++) {
      flux[li] = 0.5 * (f_L[li] + f_R[li] - diss[li]);
    }

  }
  
}
