#include "free_mem.h"


namespace block3d_cuda {

  // ---------------------------------------------------------------------------

  extern __constant__ Block3dInfo blk_info;
  
  // ---------------------------------------------------------------------------


  void free_mem(Block3dData *block_data) {

    // Release device memory 
  
    hipFree(block_data->dt);

    hipFree(block_data->xi_x);
    hipFree(block_data->xi_y);
    hipFree(block_data->xi_z);
    hipFree(block_data->eta_x);
    hipFree(block_data->eta_y);
    hipFree(block_data->eta_z);
    hipFree(block_data->zeta_x);
    hipFree(block_data->zeta_y);
    hipFree(block_data->zeta_z);

    hipFree(block_data->Jac);
  
    hipFree(block_data->rho);
    hipFree(block_data->u);
    hipFree(block_data->v);
    hipFree(block_data->w);
    hipFree(block_data->p);

#ifndef IS_INVISCID
    hipFree(block_data->T);
    hipFree(block_data->mu);

    hipFree(block_data->u_xi);
    hipFree(block_data->v_xi);
    hipFree(block_data->w_xi);
    hipFree(block_data->u_eta);
    hipFree(block_data->v_eta);
    hipFree(block_data->w_eta);
    hipFree(block_data->u_zeta);
    hipFree(block_data->v_zeta);
    hipFree(block_data->w_zeta);

    hipFree(block_data->T_xi);
    hipFree(block_data->T_eta);
    hipFree(block_data->T_zeta);

    hipFree(block_data->tau_xx);
    hipFree(block_data->tau_yy);
    hipFree(block_data->tau_zz);
    hipFree(block_data->tau_xy);
    hipFree(block_data->tau_xz);
    hipFree(block_data->tau_yz);

    hipFree(block_data->q_x);
    hipFree(block_data->q_y);
    hipFree(block_data->q_z);

    hipFree(block_data->Ev);
    hipFree(block_data->Fv);
    hipFree(block_data->Gv);

    hipFree(block_data->diff_flux_vis);
#endif

    hipFree(block_data->Q);
    hipFree(block_data->Q_p);
  
    hipFree(block_data->Ep);
    hipFree(block_data->Fp);
    hipFree(block_data->Gp);
  
  }

}
